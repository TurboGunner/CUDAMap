#include "hip/hip_runtime.h"
﻿#include ""
#include "hashmap.cuh"

#include <iostream>

#include <stdio.h>

__global__ void TestKernel(float* result, HashMap<float, float, HashFunc<float>>& map)
{
    float output = 1;
    result = &output;
}

int main()
{
    HashMap<float, float, HashFunc<float>> map;

    hipError_t cuda_status = hipSetDevice(0);

    if (cuda_status != hipSuccess) {
        fprintf(stderr, "CudaSetDevice failed!");
        return 1;
    }

    map.Put(2.0f, 2.0f);
    map.Put(3.0f, 100.0f);
    map.Put(3.0f, 120.0f);

    float* result;
    hipMalloc(&result, sizeof(float));

    TestKernel<<<1, 1>>> (result, map);


    function<hipError_t()> error_func = []() { return hipGetLastError(); };
    cuda_status = WrapperFunction(error_func, "Main", "GPUAccessLastError", cuda_status, "");

    function<hipError_t()> sync_func = []() { return hipDeviceSynchronize(); };
    cuda_status = WrapperFunction(sync_func, "Main", "GPUAccessSyncFunc", cuda_status, "");

    map.~HashMap();

    cuda_status = hipDeviceReset();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}