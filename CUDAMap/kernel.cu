#include "hip/hip_runtime.h"
#include ""
#include "hashmap.cuh"

#include <iostream>

#include <stdio.h>

__global__ void TestKernel(HashMap<float, float, HashFunc<float>>* map)
{
    printf("%f", map->Get(4.0f));
}

int main()
{
    hipError_t cuda_status = hipSuccess;

    HashMap<float, float, HashFunc<float>>* map = new HashMap<float, float, HashFunc<float>>();

    function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "Main", "GPUSetDevice", cuda_status, "");

    map->Put(2.0f, 2.0f);
    map->Put(3.0f, 100.0f);
    map->Put(4.0f, 120.0f);
    map->Put(1.0f, 1220.0f);

    TestKernel<<<1, 1>>> (map);

    function<hipError_t()> error_func = []() { return hipGetLastError(); };
    cuda_status = WrapperFunction(error_func, "Main", "GPUAccessLastError", cuda_status, "");

    function<hipError_t()> sync_func = []() { return hipDeviceSynchronize(); };
    cuda_status = WrapperFunction(sync_func, "Main", "GPUAccessSyncFunc", cuda_status, "");

    delete map;

    function<hipError_t()> reset_func = []() { return hipDeviceReset(); };
    cuda_status = WrapperFunction(reset_func, "Main", "GPUDeviceReset", cuda_status, "");

    return 0;
}
